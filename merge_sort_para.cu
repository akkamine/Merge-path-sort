
#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <stdio.h>

#define NTPB 1024

// Tableau de 12000 int max

#define N 5000
 
 __global__ void mergeSmall_k(int *A, int *B, int *M, const int sizeA, const int sizeB){
  
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   
  __shared__ int sA[N];
  __shared__ int sB[N];
  
  sA[i] = A[i];
  sB[i] = B[i];
     
  __syncthreads();
     
  int K[2];
  int P[2];
  int offset;
  int Q[2];
  
  if(i>sizeA){
    K[0] = i - sizeA;
    K[1] = sizeA;
    P[0] = sizeA;
    P[1] = i - sizeA;
  }
  else{
    K[0] = 0;
    K[1] = i;
    P[0] = i;
    P[1] = 0;
  }
  
  while(1){
    offset = abs(K[1]-P[1])/2;
    Q[0] = K[0] + offset;
    Q[1] = K[1] - offset;
    if(Q[1] >= 0 && Q[0] <= sizeB &&  (Q[1] = sizeA || Q[1] == 0 || sA[Q[1]-1] <= sB[Q[0]])){
      if(Q[0] ==sizeB || Q[1] == 0 || sA[Q[1]-1] <= sB[Q[0]]){
        if(Q[1] < sizeA && (Q[0] == sizeB || sA[Q[1]] <= sB[Q[0]])){
          M[i] = sA[Q[1]];
        }
        else{
          M[i] = sB[Q[0]];
        }
        break;
      }
      else{
        K[0] = Q[0] + 1;
        K[1] = Q[1] - 1;
      }
    }
    else{
      P[0] = Q[0] - 1;
      P[1] = Q[1] + 1;
    }
  }

}
 
int main(){
  
  int A[N];
  int B[N];
  int M[N];

  float time= 0.;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  for (int i = 0; i < N; i++) {
    A[i] = i;
    B[i] = i+1;
  }

  int *A_gpu;
  int *B_gpu;
  int *M_gpu;
  
  hipMalloc(&A_gpu, N* sizeof(int));
  hipMalloc(&B_gpu, N* sizeof(int));
  hipMalloc(&M_gpu, 2*N* sizeof(int));

  // run version with static shared memory
  hipMemcpy(A_gpu, A, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, N*sizeof(int), hipMemcpyHostToDevice);

  hipEventRecord(start);

  int nb_block = (N+NTPB-1)/NTPB;
  mergeSmall_k<<<nb_block,NTPB>>>(A_gpu, B_gpu, M_gpu, N, N);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&time, start, stop);
  printf("staticReverse: temps écoulé = %f secs\n", time/1000);

  hipMemcpy(M, M_gpu, 2*N*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < 10; i++)
    printf("M[%d] = %d\n", i, M[i]);
  
  
  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(M_gpu);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  return 0;
}
