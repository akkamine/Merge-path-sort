#include <stdio.h>

int main(void){
	// on déclare la structure qui servira de conteneur pour les informations qu'on requêtera
	hipDeviceProp_t devProp;
	// on fait notre requête au premier GPU (indice=0) pour avoir ses informations
	hipGetDeviceProperties(&devProp, 0);
	// on affiche ce qui nous intéresse dans ce qui a été récupéré
    /*
	printf("maxGridSize = (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("maxThreadsDim = (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxThreadsPerBlock = %d\n", prop.maxThreadsPerBlock);
	// voir le type de structure "hipDeviceProp_t" dans la doc pour d'autres champs...
    */
    
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ?"Yes" : "No"));
}
