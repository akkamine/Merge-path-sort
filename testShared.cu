
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void staticReverse(int *d, int n)
{
  __shared__ int s[64];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

__global__ void dynamicReverse(int *d, int n)
{
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main(void)
{
  const int n = 1024;
  int a[n], r[n], d[n];

  float time= 0.;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n-i-1;
    d[i] = 0;
  }

  int *d_d;
  hipMalloc(&d_d, n * sizeof(int));

  // run version with static shared memory
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);

  hipEventRecord(start);

  staticReverse<<<1,n>>>(d_d, n);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&time, start, stop);
  printf("staticReverse: temps écoulé = %f secs\n", time/1000);

  hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);

  // run dynamic shared memory version
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  hipEventRecord(start);

  dynamicReverse<<<1,n,n*sizeof(int)>>>(d_d, n);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&time, start, stop);
  printf("dynamicReverse: temps écoulé = %f secs\n", time/1000);

  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);
}
