
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define N 100
#define TPB 64
#define RAD 1 // radius of the stencil

__global__ void forwardCR(float *a, float *b, float *c, float*d, int k){
    
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(((i+1) % k) != 0)
        return;
    
    float k1 = a[i] / b[i-k];
    float k2 = c[i] / b[i+k];
    
    a[i] = -a[i-k] * k1;
    b[i] = b[i] - c[i-k] * k1 - a[i+k] * k2;
    
    c[i] = -c[i+k] * k2;
    d[i] = d[i] - d[i-k] * k1 - d[i+k] * k2;
}
    
__global__ void backwardCR(float *a, float *b, float *c, float*d, float *x, int k){
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(((i+1) % k) != 0)
        return;
    
    x[i] = ( d[i] - a[i] * x[i-k] - c[i] * x[i+k] ) / b[i];
}

__global__ void solve2unknown(float *a, float *b, float *c, float*d, float *x, int k){
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(((i+1) % k) != 0)
        return;
    
    x[2*k] = ( d[2*k] - a[2*k]*d[k]/b[k] ) / ( b[2*k] - a[2*k]*c[k]/b[k] );
    x[k] = (d[k] - c[k]*x[2*k]) / b[k];
    
}
    
        
    

int main(){
    srand(time(NULL));
    
    float *a_cpu, *b_cpu, *c_cpu, *d_cpu, *x_cpu;
    float *a_gpu, *b_gpu, *c_gpu, *d_gpu, *x_gpu;
    
    // a 1ere sous diagonale | b main diag | c 1ere up diag
    // d right hand vector | x solution
  	a_cpu = (float*) malloc(N*sizeof(float));
    b_cpu = (float*) malloc(N*sizeof(float));
    c_cpu = (float*) malloc(N*sizeof(float));
    d_cpu = (float*) malloc(N*sizeof(float));
    x_cpu = (float*) malloc(N*sizeof(float));
    
    hipMalloc(&a_gpu, N*sizeof(float));
    hipMalloc(&b_gpu, N*sizeof(float));
    hipMalloc(&c_gpu, N*sizeof(float));
    hipMalloc(&d_gpu, N*sizeof(float));
    hipMalloc(&x_gpu, N*sizeof(float));
    
    int i, k;
    
    for(i=0; i<N; i++){
        a_cpu[i] = (float)rand()/(float)(RAND_MAX/10);
        b_cpu[i] = (float)rand()/(float)(RAND_MAX/10);
        c_cpu[i] = (float)rand()/(float)(RAND_MAX/10);
        d_cpu[i] = (float)rand()/(float)(RAND_MAX/10);
    }
    a_cpu[0] = 0;
    c_cpu[N-1] = 0;
    
    hipMemcpy(a_gpu, a_cpu, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b_cpu, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c_cpu, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_gpu, d_cpu, N*sizeof(float), hipMemcpyHostToDevice);

    k = 2;
    while(N/k > 1){
        forwardCR<<<(N+TPB-1)/TPB, TPB>>>(a_gpu, b_gpu, c_gpu, d_gpu, k);
        hipDeviceSynchronize();
        k *= 2;
    }
    
    solve2unknown<<<(N+TPB-1)/TPB, TPB>>>(a_gpu, b_gpu, c_gpu, d_gpu, x_gpu, k);
    hipDeviceSynchronize();
    k /= 2;
    
    
    while(N/k < N){
        backwardCR<<<(N+TPB-1)/TPB, TPB>>>(a_gpu, b_gpu, c_gpu, d_gpu, x_gpu, k);
        hipDeviceSynchronize();
        k /= 2;
    }
    
    hipMemcpy(x_cpu, x_gpu, N*sizeof(float), hipMemcpyDeviceToHost);
    
    
    for(i=0; i<N; i++){
        printf("x_cpu[%d] = %f\n", i, x_cpu[i]);
    }
    
    free(a_cpu);
    free(b_cpu);
    free(c_cpu);
    free(d_cpu);
    free(x_cpu);
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
    hipFree(d_gpu);
    hipFree(x_gpu);
    
    return 0;
}
