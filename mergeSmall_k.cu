
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>


#define N 512
#define NTPB 1024

__global__ void mergeSmall_k(int *a, int *b, int *m, int sizeA, int sizeB){
	int K[2];
	int P[2];
	int Q[2];

	int i = threadIdx.x;// + blockIdx.x * blockDim.x;
	__shared__ int sA[N];
	__shared__ int sB[N];
	if(i<2*N){
		sA[i%N] = a[i%N];
		sB[i%N] = b[i%N];
		__syncthreads();

		if(i>sizeA){
			K[0] = i - sizeA;
			K[1] = sizeA;
			P[0] = sizeA;
			P[1] = i - sizeA;
		}
		else{
			K[0] = 0; K[1] = i;
			P[0] = i; P[1] = 0;
		}

		while(1){

			int offset = (K[1]-P[1])/2;
			Q[0] = K[0] + offset;
			Q[1] = K[1] - offset;

			if(Q[1] >= 0 && Q[0] <= sizeB && (Q[1] == sizeA || Q[0] == 0 || sA[ Q[1] ] > sB[ Q[0]-1 ]) ){
				if(Q[0] == sizeB || Q[1] == 0 || sA[ Q[1]-1 ] <= sB[ Q[0] ]){
					if(Q[1] < sizeA && (Q[0] == sizeB || sA[ Q[1] ] <= sB[ Q[0] ])){
		    				m[i] = sA[Q[1]] ;
		  			}
		  			else{
						m[i] = sB[Q[0]];
		  			}
		  			break;
				}
				else{
					K[0] = Q[0] + 1;
					K[1] = Q[1] - 1;
				}
			}
			else{
				P[0] = Q[0] - 1;
				P[1] = Q[1] + 1;
			}
		}
 	}
}


int main(void){
	int *A;
	int *B;
	int *M;
	//int nb_block = (N+NTPB-1)/NTPB;

	float time= 0.;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	A = (int*) malloc(sizeof(int)*N);
	B = (int*) malloc(sizeof(int)*N);
	M = (int*) calloc(sizeof(int),2*N);

	for (int i = 0; i < N; i++){
		A[i] = i*2+1;
		B[i] = i*2;
	}

	int *A_gpu;
	int *B_gpu;
	int *M_gpu;

	hipMalloc(&A_gpu, N* sizeof(int));
	hipMalloc(&B_gpu, N* sizeof(int));
	hipMalloc(&M_gpu, 2*N*sizeof(int));


	hipMemcpy(A_gpu, A, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, N*sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start);

	mergeSmall_k<<<1,NTPB>>>(A_gpu, B_gpu, M_gpu, N, N);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("mergeSmall_k: temps écoulé = %f secs\n", time/1000);


	hipMemcpy(M, M_gpu, 2*N*sizeof(int), hipMemcpyDeviceToHost);
	//for (int i = 0; i < 2*N; i++)
	//	printf("M[%d] = %d\n", i, M[i]);


	free(A);
	free(B);
	free(M);
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(M_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
